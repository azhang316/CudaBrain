#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "layer.h"
#include <iostream>

#define BLOCK_SIZE 32

__device__ void Dense(float *input, float *weights, float *output)
{
//    int x = blockIdx.x*blockDim.x+threadIdx.x;
//    int y = blockIdx.y*blockDim.y+threadIdx.y;

    
}

__device__ void Sigmoid(int *input, int* output)
{

}

__global__ void FeedForward(float data[], float weights[], /*float bias[],*/ float output[])
{
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;

    int row = blockRow*BLOCK_SIZE + threadIdx.y;
    int col = blockCol*BLOCK_SIZE + threadIdx.x;

    int pos = row * gridDim.y * BLOCK_SIZE + col;

    float Cvalue = 0;//bias[pos];

    int prevrows = row*gridDim.y*BLOCK_SIZE + threadCol;
    int prevcols = threadRow*gridDim.y*BLOCK_SIZE + col;

    //processes each output chunk one submatrix at a time to use shared memory optimally
    for(int m = 0; m < gridDim.y; m++)
    {
        //Loading submatrixes into shared memory, Bs is also transposed
        As[threadRow][threadCol] = data[prevrows + m*BLOCK_SIZE];
        Bs[threadRow][threadCol] = weights[prevcols + m*BLOCK_SIZE*gridDim.x*BLOCK_SIZE];
        __syncthreads();

        //Increments Cvalue for all the shared memory elements
    #pragma unroll
        for(int e = 0; e < BLOCK_SIZE; e++)
            Cvalue += As[threadRow][e] * Bs[e][threadCol];
        __syncthreads();
    }

    output[pos] = Cvalue;
}
/*
int fit(Dense model[], float *data, float *labels,
        const int epochs, int batch_size, int val_split)
{
    //feed forward parth, matrix multiply each input by the weights matrix
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    for(int i=0; i<sizeof(model)/sizeof(model[0]); i++)
    {
        dim3 dimGrid(model[i].data_len / BLOCK_SIZE)
        FeedForward<<<,100>>>(model[i])
    }

    //for(int i=sizeof(model)/sizeof(model[0]), );

    return 1;
}
*/


void testMatMul()
{
    int2 size = make_int2(16384,16384);
    float data[size.x * size.y];

    for(int i = 0; i < 16384*16384; i++)
        data[i] = 1;

    float* d_data;
    hipMalloc(&d_data, size.x * size.y * sizeof(float));
    hipMemcpy(d_data, data, size.x * size.y * sizeof(float), hipMemcpyHostToDevice);

    float* d_weights;
    hipMalloc(&d_weights, size.x * size.y * sizeof(float));
    hipMemcpy(d_weights, data, size.x * size.y * sizeof(float), hipMemcpyHostToDevice);
    
    float* d_output;
    hipMalloc(&d_output, size.x * size.y * sizeof(float));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(size.x/BLOCK_SIZE, 16384/BLOCK_SIZE);

        //Use Cuda Events for timing
        hipEvent_t start, stop;
        float time;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        FeedForward<<<dimGrid, dimBlock>>>(d_data, d_weights, d_output);
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        std::cout<< " Shared Memory Matrix Multiplication time =" << '\t'
                 << time << "ms" << std::endl;
}


int main()
{
    int2 size = make_int2(16384,16384);
    int labelsize = 2;
    float data[size.x * size.y];
    float labels[size.x * labelsize];

    for(int i = 0; i < 16384*16384; i++)
        data[i] = 1;

    float* d_data;
    float* d_labels;

    hipMalloc(&d_data, size.x * size.y * sizeof(float));
    hipMemcpy(d_data, data, size.x * size.y * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_labels, size.x * labelsize * sizeof(float));
    hipMemcpy(d_labels, labels, size.x * labelsize * sizeof(float), hipMemcpyHostToDevice);

    Dense l1 = Dense(d_data, 100,100, 10, 1, 1);
    Dense l2 = Dense(l1.d_output, 100,10, 1, 1, 1);

    Dense model[2] = {input, l1, l2};


    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(size.x/BLOCK_SIZE, 16384/BLOCK_SIZE);

        //Use Cuda Events for timing
        hipEvent_t start, stop;
        float time;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        FeedForward<<<dimGrid, dimBlock>>>(d_data, d_weights, d_output);
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        std::cout<< " Shared Memory Matrix Multiplication time =" << '\t'
                 << time << "ms" << std::endl;

    int epochs = 10;
    int batch_size = 100;
    int validation_split = 0.2;
    //fit(model, d_data, d_labels, 
    //    epochs, batch_size, validation_split);

}
