#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "layer.h"
#include <iostream>
#include <math.h>
#include <random>
#include <stdio.h>
#include <time.h>

#define BLOCK_SIZE 32


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ inline float sigmoid(float val)
{
    return 1.0f/ (1.0f + exp(-val));
}

// Weight initialization is done serially, as the majority of random number
// generators are sequential by nature and C random libraries cannot be used on device.
// While there are some counter based random number
// generators, the ones I found were quite difficult to understand.
// Thus I optimized this by using the Marsaglia algorithm
void initializeWeights(float *d_weights, int size)
{
    //printf("generating %d random numbers\n", size);
    //clock_t start = clock();
    
    float *rands = (float*)malloc(size*sizeof(float));

    /* no longer using c++ default rng
    std::default_random_engine generator;
    std::normal_distribution<float> distribution{0.0,0.01};
    for (int i=0; i<size; i++)
        rands[i] = distribution(generator); */

// Implementing the Marsaglia polar method of generating random numbers on
// a normal distribution. The standard deviation that is used is 0.01
    float r1 = 1, r2 = 1, S = 1, root;
    for(int i = 0; i<size-1; i+=2)
    {
        do
        {
            //too bad rand() doesnt work on gpu device ... 
            r1 = 2.0 * rand() / (double)RAND_MAX - 1;
            r2 = 2.0 * rand() / (double)RAND_MAX - 1;
            S = r1*r1 + r2*r2;
        }while(S >=1 || S==0);
        root = sqrt(-2 * log(S) / S)/100; //sqrt(-2.0 * log(S) / S) * 0.01;
        rands[i] = r1 * root;
        rands[i+1] = r2 * root;
    }
    rands[size-1] = r2 * root;

    hipMemcpy(d_weights, rands, size*sizeof(float), hipMemcpyHostToDevice);

    //clock_t end = clock();
    //double cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    //printf("initialize weights took %f seconds \n", cpu_time_used);
    //for(int i = 0; i < size; i++)
    //    printf("%f\n",rands[i]);
}

__global__
void matTran(float *A, float *B)
{
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    int row = threadIdx.y;
    int col = threadIdx.x;

    int pos = blockRow*BLOCK_SIZE*BLOCK_SIZE+row*BLOCK_SIZE + 
          blockCol*BLOCK_SIZE+ col;
    As[col][row] = A[pos];
    __syncthreads();

    B[pos] = As[row][col];
}

// expects weights to be transposed already
__global__ void feedForward(float data[], float weights[], float bias[], float output[], dim3 size, int activation)
{
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;
    
    int row = blockRow*BLOCK_SIZE + threadRow;
    int col = blockCol*BLOCK_SIZE + threadCol;

    if(row > size.x || col > size.z)
        return;

    int pos = row * size.z + col;

    float Cvalue = bias[col];
    
    int aprevrows = row*size.z + threadCol;
    int bprevrows = col*size.z + threadCol;

    //processes each output chunk one submatrix at a time to use shared memory optimally
    for(int m = 0; m < gridDim.y; m++)
    {
        //Loading submatrixes into shared memory, Bs is also transposed
        As[threadRow][threadCol] = data[aprevrows + m*BLOCK_SIZE];
        Bs[threadRow][threadCol] = weights[bprevrows + m*BLOCK_SIZE];
        __syncthreads();

        //Increments Cvalue for all the shared memory elements
    #pragma unroll
        for(int e = 0; e < BLOCK_SIZE; e++)
            Cvalue += As[threadRow][e] * Bs[threadCol][e];
        __syncthreads();
    }
    if(activation == SIGMOID)
        output[pos] = sigmoid(Cvalue);
    else
        output[pos] = Cvalue;
}

__device__ void mse(float *d_prediction, float *d_actual, float *out)
{
    extern __shared__ float sdata[];
    
    int myId = threadIdx.x + blockDim.x*blockIdx.x;
    int tid = threadIdx.x;
    sdata[tid] = (d_prediction[myId] - d_actual[myId]);
    sdata[tid] *= sdata[tid];

    __syncthreads();

    for(int s = blockDim.x/2; s>0; s>>=1)
    {
        if(tid<s)
            sdata[tid] += sdata[tid+s];
        __syncthreads();
    }
    
    if(tid ==0)
        out[blockIdx.x] = sdata[0];    
}

__device__ float deriv_error(float d_output, float d_actual, float d_weights )
{
    //float de_dout = d_output - d_actual; //previous derivative error
    //float dout_dnet = d_output[] * (1-output[i]);
    //de_dout * dout_dnet; // * sum(weights);
    return 1.0f;
} 

__device__ float gradient(float *d_input, float *d_output,float *deriv_error) //used for changing weight values in update
{
    //float de_dout = deriv_error(); // d_output[] - d_actual;
    //float dout_dnet = d_output[] * (1-d_output[]);
    //float dnet_dweight = d_input[]; // can be adapted to bias term by making this 1.

    //return = de_dout * dout_dnet * dnet_dweight;
    return 1.0f;
}

__device__ void update()
{
    //d_weight -= learning_rate * gradient()
}

// implementing a a map operation followed by a gather operation 
// the gather operation is only used on each column, while
// the rows are left separate.
__global__ void derivativeError(float *output, float *actual, float *deriv_err)
{
    __shared__ float sdata[1024];

    //ideally block is 1024x1 and grid is ??? x units
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y;
    
    sdata[threadIdx.x] = output[row*gridDim.y + col];
    __syncthreads();

    for(int s= blockDim.x / 2; s>0; s>>=1)
    {
        if(threadIdx.x < s)
            sdata[threadIdx.x] += sdata[threadIdx.x+s];
        __syncthreads();
    }
    if(threadIdx.x == 0) //only tid0 can write
    {    
        /*deriv_err[blockIdx.x] = sdata[0]*/deriv_err[blockDim.y*blockIdx.x+col] = sdata[blockIdx.x];
    }
}

__global__ void backPropagate(float *deriv_err, float *prev_deriv_err, 
                              float *wieghts, float *output)
{
    //use map operation to multiply d_output[i]*(1-output[i])*prev_deriv_error[i]*weight[i]
    //use gather operation to gather all these together.
}

int fit(Dense model[], float *data, float *labels,
        const int num_layers, const int epochs, int batch_size, int val_split)
{
    float* d_data;
    float* d_labels;
    int last = num_layers - 1;
    int data_size = model[0].size.x * model[0].size.y * sizeof(float);
    int label_size = model[last].size.x * model[last].size.z * sizeof(float);
    hipMalloc(&d_data, data_size);
    hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice);
    hipMalloc(&d_labels, label_size);
    hipMemcpy(d_labels, labels, label_size, hipMemcpyHostToDevice);

    
    model[0].d_data = d_data;

    printf("Randomly initializing weights ...");
    for(int i=0; i<num_layers; i++)
    {
        initializeWeights(model[i].d_weights, model[i].size.y * model[i].size.z);
        initializeWeights(model[i].d_bias, model[i].size.z);
    }
    printf("Done.\n");

    //feed forward parth, matrix multiply each input by the weights matrix
    for(int i=0; i<num_layers; i++)
    {
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid((model[i].size.x - 1) / BLOCK_SIZE + 1,
                     (model[i].size.z - 1) / BLOCK_SIZE + 1);
        printf("feedforward gridsize: %d,%d\n", dimGrid.x, dimGrid.y);
        feedForward<<<dimGrid, dimBlock>>>(model[i].d_data, model[i].d_weights, 
                model[i].d_bias, model[i].d_output, 
                model[i].size, model[i].activation);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }

    dim3 dimBlock1(1024,1);
    dim3 dimGrid1((model[last].size.y-1)/1024+1,model[last].size.z);
    derivativeError<<<dimGrid1, dimBlock1>>>(model[last].d_output, 
                      d_labels, model[last].deriv_error);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
   
    
    //derivativeError<<<1, dimBlock1>>>(model[last].deriv_error, 0,
      //                                   d_labels, model[last].deriv_error);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    
    float *error = (float*) malloc(label_size);
    hipMemcpy(error, model[last].deriv_error, label_size, hipMemcpyDeviceToHost);

    for(int i=0; i< 256; i++)
        printf("%f \n", error[i]);

    return 1;
}


void testMatMul(int s)
{
    int2 size = make_int2(s,s);
    float data[size.x * size.y];

    for(int i = 0; i < s*s; i++)
        data[i] = 1;

    float* d_data;
    hipMalloc(&d_data, size.x * size.y * sizeof(float));
    hipMemcpy(d_data, data, size.x * size.y * sizeof(float), hipMemcpyHostToDevice);

    float* d_weights;
    hipMalloc(&d_weights, size.x * size.y * sizeof(float));
    hipMemcpy(d_weights, data, size.x * size.y * sizeof(float), hipMemcpyHostToDevice);
    
    float* d_output;
    hipMalloc(&d_output, size.x * size.y * sizeof(float));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(size.x/BLOCK_SIZE, size.y/BLOCK_SIZE);

        //Use Cuda Events for timing
        hipEvent_t start, stop;
        float time;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        float *transposed;
        hipMalloc(&transposed, size.x * size.y * sizeof(float));
        matTran<<<dimGrid, dimBlock>>>(d_weights, transposed);
        feedForward<<<dimGrid, dimBlock>>>(d_data, transposed, d_output, d_output, dim3(s,s,s), SIGMOID);
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        std::cout<< " Shared Memory Matrix Multiplication time =" << '\t'
                 << time << "ms" << std::endl;
}

int main()
{
    printf("start");
    int2 size = make_int2(256,256);
    int labelsize = 2;
    float *data = (float*)malloc(size.x * size.y * sizeof(float));
    float *labels = (float*)malloc(size.x * labelsize * sizeof(float));

    for(int i = 0; i < 256*256; i++)
        data[i] = 1;

    float* d_data;
    float* d_labels;

    printf("data size: %d ... inputted %d\n", sizeof(data), size.x * size.y * sizeof(float));

    hipMalloc(&d_data, size.x * size.y * sizeof(float));
    hipMemcpy(d_data, data, size.x * size.y * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_labels, size.x * labelsize * sizeof(float));
    hipMemcpy(d_labels, labels, size.x * labelsize * sizeof(float), hipMemcpyHostToDevice);

    printf("before creating dense layers \n");

    Dense l1 = Dense(d_data, 256, 256, 64, SIGMOID, 1); 
    Dense l2 = Dense(l1.d_output, 256, 64, 32, SIGMOID, 1);
    Dense l3 = Dense(l2.d_output, 256, 32, 16, SIGMOID, 1);
    const int num_layers = 3;
    
    Dense model[num_layers] ={l1, l2, l3};

    int epochs = 10;
    int batch_size = 100;
    int validation_split = 0.2;
    fit(model, data, labels, 
        num_layers, epochs, batch_size, validation_split);

    /*
    testMatMul(16);
    testMatMul(16);
    testMatMul(64);
    testMatMul(256);
    testMatMul(2048);
    testMatMul(8192);
    testMatMul(16384);
    testMatMul(32768);*/
}
